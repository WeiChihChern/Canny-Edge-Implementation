#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>

#include "hipblas.h"
#include "Canny_Cuda.cuh"
#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include ""


using namespace std;

#define num_threads 16



// Should only allocate 1D blocks with threads
template <typename inputType>
__global__ void conv2_h_sobel_cuda_function(
	const inputType* data, float* dst, int rows, int cols, float* kernel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int cur_row = idx / cols;
	int cur_col = idx % cols;
	
	if (cur_col > 0 && cur_col < cols - 1)
		dst[idx] = data[idx - 1]  * *kernel + data[idx]     * *(kernel+1) + data[idx + 1] * *(kernel+2);

	return;
}


template <typename inputType>
__global__ void conv2_v_sobel_cuda_function(
	const inputType *data, float *dst, int rows, int cols, float* kernel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int cur_row = idx / cols;
	int cur_col = idx % cols;

	if (cur_row > 0 && cur_row < rows - 1)
		dst[idx] = data[idx - cols]  * *kernel + data[idx] * *(kernel + 1) + data[idx + cols] * *(kernel + 2);

	return;
}



// This magnitude function will store the result in gx
template <typename inputType>
__global__ void calculate_magnitude_cuda_function(
	const inputType *gy, const inputType *gx, inputType *dst, int rows, int cols)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int cur_row = idx / cols;
	int cur_col = idx % cols;

	if (cur_col > 0 && cur_col < cols - 1 && 
		cur_row > 0 && cur_row < rows - 1)
		dst[idx] = std::sqrt(gy[idx] + gx[idx]);

	return;
}



template <typename inputType>
__global__ void gradient_cuda_function(const inputType* gx, const inputType* gy, int rows, int cols, inputType* dst)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int cur_row = idx / cols;
	int cur_col = idx % cols;

	if (cur_col > 0 && cur_col < cols - 1 && 
		cur_row > 0 && cur_row < rows - 1)
	{
			float w = abs(gy[idx] / (gx[idx]+0.0001));

			if (w < 0.4)
				dst[idx] = 0;
			else if (w > 2.3)
				dst[idx] = 90;
			else 
				dst[idx] = 45;
	}

}





template <typename inputType>
__global__ void nonMax_cuda_function(
	const inputType* magnitude, const inputType* gradient, 
	const inputType *gx, const inputType *gy, inputType* dst, 
	int rows, int cols, float high_thres, float low_thres)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int cur_row = idx / cols;
	int cur_col = idx % cols;

	inputType cur_magnitude, theta;


	if (cur_col > 0 && cur_col < cols - 1 && 
		cur_row > 0 && cur_row < rows - 1)
	{
		cur_magnitude = magnitude[idx];
		theta         = gradient[idx];

		if ( cur_magnitude > low_thres && cur_magnitude != 0 ) // Edge pixel
		{ 
				if (theta == 90) 
				{
						// vertical direction
						if ( cur_magnitude > magnitude[idx - cols] && cur_magnitude >= magnitude[idx + cols] ) 
							dst[idx] = (cur_magnitude >= high_thres) ? 255 : cur_magnitude;
				}
				else if (theta == 0) 
				{
						// horizontal direction
						if (cur_magnitude > magnitude[idx - 1] && cur_magnitude >= magnitude[idx + 1]) 
							dst[idx] = (cur_magnitude >= high_thres) ? 255 : cur_magnitude;
				}
				else  if (theta == 45)// bottom-left to top-right  or  bottom-right to top-left direction
				{ 
						int d = (gy[idx] * gx[idx] < 0) ? 1 : -1;
						if (cur_magnitude >= magnitude[idx + cols - d] && cur_magnitude > magnitude[idx - cols + d]) 
							dst[idx] = (cur_magnitude >= high_thres) ? 255 : cur_magnitude;
				}
				else 
							dst[idx] = 0;
		}
		else // Non edge pixel
				dst[idx] = 0; 
	}
}



__global__ void normalizedTo(
	float *src, size_t n, int max_idx, int min_idx, float range, float range2, float high_val,
	float low_val)
{
	float min_val = src[min_idx];


	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx >= 0 && idx < n)
		src[idx] = ((src[idx] - min_val) / range) * range2 + low_val;
	
}



template <typename inputType>
__global__ void hysteresis_cuda_function(
	inputType* src, int rows, int cols, float high_thres, float low_thres)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int cur_row = idx / cols;
	int cur_col = idx % cols;


	if (cur_col > 0 && cur_col < cols - 1 && 
		cur_row > 0 && cur_row < rows - 1)
	{
			if(src[idx] < high_thres && src[idx] > low_thres)
			{
						if (*(src + idx - 1)        == 255 || *(src + idx + 1)        == 255 || *(src+idx - cols) == 255 || 
							*(src + idx + cols)     == 255 || *(src + idx - cols - 1) == 255 || 
							*(src + idx - cols + 1) == 255 || *(src + idx + cols + 1) == 255 || *(src + idx + cols - 1) == 255) 
						{
							src[idx] = 255;
						}
						else // No strong pixel (=255) in 8 neighbors
						{ 
							src[idx] = 0;
						}
			}	
	}
}



template <typename inputType>
__global__ void makeZero(inputType* src, int rows, int cols)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int cur_row = idx / cols;
	int cur_col = idx % cols;

	if(cur_col >= 0 && cur_col < cols &&
	   cur_row >= 0 && cur_row < rows)
	   src[idx] = 0;

}













extern "C"
void conv2_sobel_cuda(unsigned char* src, float* dst, int r, int c, float high_thres, float low_thres)
{

	// Initializing
	int	size = r*c;

	kernel kernel;

	unsigned char* gpu_src;
	float 	
		*gpu_dst, *gpu_dst_gy, *gpu_dst_gx,
		*gpu_kernel1, *gpu_kernel2,
		*gpu_magnitude, *gpu_gradient,
		*gpu_nonMax;
	hipError_t  error, cudaStatus;


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}





	//
	//
	// Allocating memory on gpu, and copy cpu memory to gpu
	error = hipMalloc((void**)&gpu_src, size * sizeof(unsigned char));
	if (error != hipSuccess) {
		cout << "hipMalloc failed\n";
		goto Error;
	}
	error = hipMalloc((void**)&gpu_dst, size * sizeof(float));
	if (error != hipSuccess) {
		cout << "hipMalloc failed\n";
		goto Error;
	}
	error = hipMalloc((void**)&gpu_dst_gy, size * sizeof(float));
	if (error != hipSuccess) {
		cout << "hipMalloc failed\n";
		goto Error;
	}
	error = hipMalloc((void**)&gpu_dst_gx, size * sizeof(float));
	if (error != hipSuccess) {
		cout << "hipMalloc failed\n";
		goto Error;
	}
	error = hipMalloc((void**)&gpu_kernel1, 3 * sizeof(float));
	if (error != hipSuccess) {
		cout << "hipMalloc failed\n";
		goto Error;
	}
	error = hipMalloc((void**)&gpu_kernel2, 3 * sizeof(float));
	if (error != hipSuccess) {
		cout << "hipMalloc failed\n";
		goto Error;
	}
	error = hipMalloc((void**)&gpu_magnitude, size * sizeof(float));
	if (error != hipSuccess) {
		cout << "hipMalloc failed\n";
		goto Error;
	}
	error = hipMalloc((void**)&gpu_gradient, size * sizeof(float));
	if (error != hipSuccess) {
		cout << "hipMalloc failed\n";
		goto Error;
	}
	error = hipMalloc((void**)&gpu_nonMax, size * sizeof(float));
	if (error != hipSuccess) {
		cout << "hipMalloc failed\n";
		goto Error;
	}
	



	//
	//
	// Copy data from host to device
	error = hipMemcpy(gpu_src, src, size * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		cout << "cudaMemcopy failed\n";
		goto Error;
	}
	error = hipMemcpy(gpu_kernel1, kernel.k1, 3 * sizeof(float), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		cout << "cudaMemcopy failed\n";
		goto Error;
	}
	error = hipMemcpy(gpu_kernel2, kernel.k2, 3 * sizeof(float), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		cout << "cudaMemcopy failed\n";
		goto Error;
	}
	


	//
	//
	// cuda functions were written in taking 1D blocks
	// so don't create a 2D blocks
	dim3 threadsPerBlock(num_threads, 1);
	dim3 blocksPerGrid(size / num_threads + 1, 1);

	// Get gy
	conv2_h_sobel_cuda_function<<< blocksPerGrid, threadsPerBlock>>>(gpu_src,  gpu_dst,   r, c, gpu_kernel1);
	conv2_v_sobel_cuda_function<<< blocksPerGrid, threadsPerBlock>>>(gpu_dst, gpu_dst_gy, r, c, gpu_kernel2);


	// Get gx
	conv2_h_sobel_cuda_function<<<blocksPerGrid, threadsPerBlock>>>(gpu_src,  gpu_dst,   r, c, gpu_kernel2);
	conv2_h_sobel_cuda_function<<<blocksPerGrid, threadsPerBlock>>>(gpu_dst, gpu_dst_gx, r, c, gpu_kernel1);


	// Get magnitude result
	makeZero<<<blocksPerGrid, threadsPerBlock>>>(gpu_magnitude, r, c);
	calculate_magnitude_cuda_function<<<blocksPerGrid, threadsPerBlock>>>(gpu_dst_gy, gpu_dst_gx, gpu_magnitude, r, c);


	int max_idx, min_idx;
	hipblasHandle_t handle;
	hipblasIsamax(handle, size, gpu_magnitude, 1, &max_idx);
	hipblasIsamin(handle, size, gpu_magnitude, 1, &min_idx);
	
	float range = src[max_idx] - src[min_idx];
	
	normalizedTo<<<blocksPerGrid, threadsPerBlock>>>(gpu_magnitude, size, max_idx, min_idx, range, 255, 255, 0);

	
	
	// Get gradient result
	// gradient_cuda_function<<<blocksPerGrid, threadsPerBlock>>>(gpu_dst_gx, gpu_dst_gy, r, c, gpu_gradient);



	// Get nonMax result
	// nonMax_cuda_function<<<blocksPerGrid, threadsPerBlock>>>(
	// 	gpu_magnitude, gpu_gradient, gpu_dst_gx, gpu_dst_gy, gpu_nonMax, r, c, high_thres, low_thres);

	

	// Get threshold result
	//hysteresis_cuda_function<<<blocksPerGrid, threadsPerBlock>>>(gpu_nonMax, r, c, high_thres, low_thres);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}



	
	// Copy the result back to cpu 
	error = hipMemcpy(dst, gpu_magnitude, size * sizeof(float), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
		cout << "cudaMemcopy failed\n";
		goto Error;
	}



Error:
	hipFree(gpu_dst);
	hipFree(gpu_src);
	hipFree(gpu_kernel1);
	hipFree(gpu_kernel2);


	hipFree(gpu_dst_gy);
	hipFree(gpu_dst_gx);
	hipFree(gpu_magnitude);
	hipFree(gpu_gradient);
	hipFree(gpu_nonMax);


	return;
}














