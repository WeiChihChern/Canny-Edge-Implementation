#include "hip/hip_runtime.h"
#pragma once


#include "Edge_Cuda.cuh"
#include "Cuda_helper.cuh"
#include <math.h>





// hipReadModeNormalizedFloat = read elements in normalized float
// hipReadModeElementType     = Read texture as specified element type
texture <float, hipTextureType2D, hipReadModeElementType>     ker_h_tex;
texture <float, hipTextureType2D, hipReadModeElementType>     ker_v_tex;
texture <uchar, hipTextureType2D, hipReadModeNormalizedFloat>		canny_tex;

hipArray *d_canny_src = 0;   // input data, will be binded to texture
hipArray *d_ker_h = 0; // input kernel, will be binded to texture
hipArray *d_ker_v = 0; // input kernel, will be binded to texture

float sobel_horizontal[9] = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };
float sobel_vertical[9] = { -1, -2, -1, 0, 0, 0, 1, 2, 1 };

float *gpu_dst_v, *gpu_dst_h;
uchar *non_max;







__global__
void conv2_h(
	float *dst, 
	int rows, 
	int cols,
	int ker_rows, 
	int ker_cols)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;

	if (x < cols && y < rows)
	{
		float sum = 0.0f;

		#pragma unroll
		for (int i = -ker_cols / 2; i < ker_cols / 2 + 1; i++)
			for (int j = -ker_rows / 2; j < ker_rows / 2 + 1; j++)
				sum += tex2D(canny_tex, x + i, y + j) * tex2D(ker_h_tex, i + 1, j + 1);
			
		dst[y*cols + x] = sum * 255; // Scale back to uchar's range
	}
};


__global__
void conv2_v(
	float *dst,
	int rows,
	int cols,
	int ker_rows,
	int ker_cols)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;

	if (x < cols && y < rows)
	{
		float sum = 0.0f;

		#pragma unroll
		for (int i = -ker_cols / 2; i < ker_cols / 2 + 1; i++)
			for (int j = -ker_rows / 2; j < ker_rows / 2 + 1; j++)
				// tex value will be normalized to 0.0~1.0 in float
				sum += tex2D(canny_tex, x + i, y + j) * tex2D(ker_v_tex, i + 1, j + 1);

		dst[y*cols + x] = sum * 255; // Scale back 
	}
};



__device__ __forceinline__
float get_magnitude(float src1, float src2)
{
	return sqrt(src1 * src1 + src2 * src2);
};

__device__ __forceinline__
float get_edgeDrections(float gy, float gx)
{
	float w = abs(gy / (gx + 0.0001));

	if (w < 0.4)
		return 0.0f;
	else if (w > 2.3)
		return 90.0f;
	else
		return (gx*gy > 0) ? -45.0f : 45.0f; 
};

__global__
void get_info_from_edge(float* src1, float* src2, int rows, int cols)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;

	int idx = y*cols + x;
	if (x < cols && y < rows)
	{

		// Temporary store results, since second calculation needs both src1, src2 untouched
		float temp_mag = get_magnitude(src1[idx], src2[idx]);
		float temp_dir = get_edgeDrections(src1[idx], src2[idx]);

		// using src1 to store magnitude result & src2 to store direction result
		src1[idx] = temp_mag;
		src2[idx] = temp_dir;
	}
};


// Cuda store 2D data in column major order
// Opencv does the otherwise
__global__
void nonMax(
	float* mag, float* gra, uchar* dst,
	int rows, int cols,
	float h_thres, float l_thres)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;


	int idx = y*cols + x;
	if (x > 1 && x < cols && y < rows && y > 1)
	{
		float cur_mag_val = mag[idx];
		float direction = gra[idx];
		if (cur_mag_val > l_thres && cur_mag_val != 0)
		{
			if (direction == 90)
			{
				if (cur_mag_val > mag[idx - 1] && cur_mag_val >= mag[idx + 1])
					dst[idx] = (cur_mag_val >= h_thres) ? 255 : 125;
				
			}
			else if (direction == 0)
			{
				if (cur_mag_val > mag[idx - cols] && cur_mag_val >= mag[idx + cols])
					dst[idx] = (cur_mag_val >= h_thres) ? 255 : 125;
			}
			else
			{
				int d = (direction == 45) ? 1 : -1;
				if (cur_mag_val >= mag[idx + cols - d] && cur_mag_val > mag[idx - cols + d])
					dst[idx] = (cur_mag_val >= h_thres) ? 255 : 125;
			}
		}
		else
			dst[idx] = 0;
	}
}








__host__ __forceinline__
void src_kernel_init(uchar *src, int rows, int cols)
{
	uint     size = cols * rows * sizeof(uchar);
	uint ker_size = 3 * 3 * sizeof(float);

	// allocate array and copy image data
	hipChannelFormatDesc channelDesc     = hipCreateChannelDesc(8,  0, 0, 0, hipChannelFormatKindUnsigned);
	hipChannelFormatDesc channelDesc_ker1 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc_ker2 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	gpuErrchk(hipMallocArray(&d_canny_src, &channelDesc, cols, rows)); // src image
	gpuErrchk(hipMallocArray(&d_ker_h, &channelDesc_ker1, 3, 3)); // kernel horizontal
	gpuErrchk(hipMallocArray(&d_ker_v, &channelDesc_ker2, 3, 3)); // kernel horizontal

	gpuErrchk(hipMemcpyToArray(d_canny_src, 0, 0, src, size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToArray(d_ker_h, 0, 0, sobel_horizontal, ker_size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToArray(d_ker_v, 0, 0, sobel_vertical, ker_size, hipMemcpyHostToDevice));

}



__host__ __forceinline__ 
void textureSetUp()
{

	// set texture parameters
	canny_tex.addressMode[0] = hipAddressModeMirror;
	canny_tex.addressMode[1] = hipAddressModeMirror;
	canny_tex.filterMode = hipFilterModePoint;      // no interpolation between pixels like pixel at (1.5, 1.5)
	canny_tex.normalized = false;                    // access texture with not-normalized coordinates
	gpuErrchk(hipBindTextureToArray(canny_tex, d_canny_src));

	ker_h_tex.addressMode[0] = hipAddressModeMirror;
	ker_h_tex.addressMode[1] = hipAddressModeMirror;
	ker_h_tex.filterMode = hipFilterModePoint;      // no interpolation between pixels like pixel at (1.5, 1.5)
	ker_h_tex.normalized = false;                    // access texture with not-normalized coordinates
	gpuErrchk(hipBindTextureToArray(ker_h_tex, d_ker_h));

	ker_v_tex.addressMode[0] = hipAddressModeMirror;
	ker_v_tex.addressMode[1] = hipAddressModeMirror;
	ker_v_tex.filterMode = hipFilterModePoint;      // no interpolation between pixels like pixel at (1.5, 1.5)
	ker_v_tex.normalized = false;                    // access texture with not-normalized coordinates
	gpuErrchk(hipBindTextureToArray(ker_v_tex, d_ker_v));

}


__host__ __forceinline__
void memClean()
{
	hipUnbindTexture(canny_tex);
	hipUnbindTexture(ker_h_tex);
	hipUnbindTexture(ker_v_tex);
	hipFreeArray(d_canny_src);
	hipFreeArray(d_ker_h);
	hipFreeArray(d_ker_v);
	hipFree(gpu_dst_v);
	hipFree(gpu_dst_h);
	hipFree(non_max);
}




extern "C"
void canny_cuda_impl(
	uchar*src, uchar *dst,
	int rows, int cols,
	float high_thres, float low_thres)
{
	// hipMallocArray & hipMemcpyToArray for input src & sobel kernels
	src_kernel_init(src, rows, cols); 
	// bind src & kernels to textures
	textureSetUp();

	// Create gpu memory to store some result
	int s_float = rows*cols * sizeof(float);
	int s_uchar = rows*cols * sizeof(uchar);
	gpuErrchk(hipMalloc((void **)&gpu_dst_v, s_float)); // store sobel vertical convolution result
	gpuErrchk(hipMalloc((void **)&gpu_dst_h, s_float)); // store sobel horizontal convlution result
	gpuErrchk(hipMalloc((void **)&non_max, s_uchar));   // store non max suppression result in uchar 


	dim3 threadPerBlock(8, 8);
	dim3 blocksPerGrid((cols / threadPerBlock.x) + 1, (rows / threadPerBlock.y) + 1);

	// Convolution with 2 sobel kernels
	conv2_h <<<blocksPerGrid, threadPerBlock>>> (gpu_dst_h, rows, cols, 3, 3);
	conv2_v <<<blocksPerGrid, threadPerBlock>>> (gpu_dst_v, rows, cols, 3, 3);



	// Bind gpu_dst_h & _v to texture




	// Get magnitude and direction result from 2 edge maps
	//	Then reuse the variable:
	//		gpu_dst_h will be replaced with magnitdue data
	//		gpu_dst_v will be replaced with direction data
	get_info_from_edge <<<blocksPerGrid, threadPerBlock>>>(gpu_dst_h, gpu_dst_v, rows, cols);

	// Performance non maximum suppression & leave the hysteresis thresholding to cpu for now
	nonMax<<<blocksPerGrid, threadPerBlock >> >(gpu_dst_h, gpu_dst_v, non_max, rows, cols, high_thres, low_thres);


	gpuErrchk(hipMemcpy(dst, non_max, s_uchar, hipMemcpyDeviceToHost));


	memClean();

	return;
}




#endif // !_EDGE_CUDA_CU_
